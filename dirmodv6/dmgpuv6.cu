// Directional modulation on GPU
// PRELIMINARY / PROOF OF CONCEPT

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_complex.h>

#define PI 3.141592653589793238462643383279502884197169399375105820974

#include <sys/time.h>

typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

void printMatrix(int m, int n, const hipComplex *A, int lda, const char *name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            hipComplex Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f%+fi\n", name, row+1, col+1, hipCrealf(Areg), hipCimagf(Areg));
        }
    }
}

// in MATLAB, the construction looked sort of like this
// for a problem with 2 specified directions:
//
//    1        (             0 0          ( Td[0]  Td[1] ) )
// ______ * exp( -2*j*pi*d * 1 1   .*  cos( Td[0]  Td[1] ) )
// sqrt(M)     (             ...          (     ...      ) )
//             (             M-1          ( Td[0]  Td[1] ) )
//
// note that the number of antenna elements determines the number
// of rows, while the number of specified directions determines the
// number of columns.
//
// the argument to exp is real until it is multiplied by j,
// which is what makes the overall result complex, so it seems
// splitting this up by Euler's formula might be the way to go
// to actually build the complex values in the last step using
// all real arithmetic up to that.
//
// the following is a first pass at what that looks like -
// it will need to be verified...
//
// the real part of H(Theta_d) =
//
//    1        (          0 0          ( Td[0]  Td[1] ) )
// ______ * cos( 2*pi*d * 1 1   .*  cos( Td[0]  Td[1] ) )
// sqrt(M)     (          ...          (     ...      ) )
//             (          M-1          ( Td[0]  Td[1] ) )
//
// the imaginary part of H(Theta_d) =
//
//   -j        (          0 0          ( Td[0]  Td[1] ) )
// ______ * sin( 2*pi*d * 1 1   .*  cos( Td[0]  Td[1] ) )
// sqrt(M)     (          ...          (     ...      ) )
//             (          M-1          ( Td[0]  Td[1] ) )
//

const unsigned int STEERING_BLOCK_SIZE = 16;

__global__ void steering(unsigned int M, unsigned int K, float d, const float *Td, hipComplex *H_Td) {

    // calculate the steering matrix
    // one element per thread, owner writes

    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    float norm = 1 / sqrtf(M);
    if ((r < M) && (c < K)) {
        float arg = 2 * PI * d * r * cosf(Td[c]);
        unsigned int cmaj = c * M + r;
        sincosf(arg, &H_Td[cmaj].y, &H_Td[cmaj].x);
        H_Td[cmaj].x *= norm;
        H_Td[cmaj].y *= -norm;
    }
}

const unsigned int CIDENT_BLOCK_SIZE = 16;

__global__ void Cident(unsigned int n, hipComplex *I) {

    // calculate a complex identity matrix
    // one element per thread, owner writes

    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;
    if ((r < n) && (c < n))
        I[c * n + r] = make_hipFloatComplex((r == c), 0.);
}

// there were pretty bad problems that seemed to be related to effective
// address calculation of structure members when the structure was on
// the device, but it was found that the cublas pointer mode needed to
// be changed...  that might have been the whole issue - there is a thought
// that the scalars had to be on the device to get async dispatches...
// not sure about that now either

// this is constant until/unless number/direction of receivers change
// if/when these changes occur, only Td_k and/or Td_deg are affected
//struct dm_parms {
//    unsigned int M;      // number of antenna elements
//    float d;    // spacing of antenna elements, in wavelengths
//    float P;    // average power, presumably Watts, typically 1 for sims
//    float B;    // beta 1; power allocation; 1 if not using null space
//    float Td_c; // pi / 180 for scaling
//    unsigned int Td_k;   // number of Td elements used, i.e., # of directions
//    float Td_deg[64]; // specified directions in degrees; Theta_d
//};

int main(int argc, char *argv[])
{
    Timer timer;

    hipStream_t cudaStream = NULL;

    hipError_t cudaError1 = hipSuccess;
    hipError_t cudaError2 = hipSuccess;
    hipError_t cudaError3 = hipSuccess;
    hipError_t cudaError4 = hipSuccess;

    hipblasHandle_t cublasHandle = NULL;
    hipblasStatus_t hipblasStatus_t = HIPBLAS_STATUS_SUCCESS;

    hipsolverHandle_t cusolverDnHandle = NULL;
    hipsolverStatus_t cusolverStatus = HIPSOLVER_STATUS_SUCCESS;

//    struct dm_parms *dm_h;
    unsigned int M;    // # of antenna elements
    unsigned int Td_k; // # of Td elements used, i.e., # of directions
    float Td_deg[64];  // specified directions in degrees; Theta_d
    float d;    // spacing of antenna elements, in wavelengths
    float P;    // average power, presumably Watts, typically 1 for sims
    float B;    // beta 1; power allocation; 1 if not using null space
    float Td_c; // pi / 180 for scaling
    hipComplex alpha, beta, scale;

    hipComplex *H_Td = NULL;  // for diagnostics
    hipComplex *HH   = NULL;
    hipComplex *LU   = NULL;
    hipComplex *I    = NULL;
    hipComplex *X    = NULL;
    hipComplex *Hdag = NULL;
    hipComplex *s    = NULL;
    hipComplex *W    = NULL;
    int *Ipiv = NULL;
    int *info = NULL;

//    struct dm_parms *dm_d;
    unsigned int *M_d;
    unsigned int *Td_k_d;
    float *Td_deg_d;
    float *d_d;
    float *P_d;
    float *B_d;
    float *Td_c_d;
    hipComplex *alpha_d, *beta_d, *scale_d;

    float *Td_d = NULL;
    hipComplex *H_Td_d = NULL;
    hipComplex *HHLU_d = NULL;
    hipComplex *IX_d   = NULL;
    hipComplex *work_d = NULL;
    hipComplex *Hdag_d = NULL;
    hipComplex *s_d  = NULL;
    hipComplex *W_d  = NULL;
    int *Ipiv_d = NULL;
    int *info_d = NULL;

    int lwork = 0;     /* size of workspace */

    hipFree(0);
    startTime(&timer);

    // set up test case
//    dm_h      = (struct dm_parms *) malloc(sizeof(struct dm_parms));
    M = 2;
    d = 0.5;
    P = 1.0;
    B = 1.0;
    Td_c = PI / 180.0;
    Td_k = 2;
    Td_deg[0] = 45.0;
    Td_deg[1] = 120.0;
    alpha = make_hipFloatComplex(1., 0.);
    beta  = make_hipFloatComplex(0., 0.);
    scale = make_hipFloatComplex(B * sqrtf(P) / sqrtf(Td_k), 0.);

    // host allocations
    H_Td = (hipComplex *) malloc(sizeof(hipComplex) * M    * Td_k);
    HH   = (hipComplex *) malloc(sizeof(hipComplex) * Td_k * Td_k);
    LU   = (hipComplex *) malloc(sizeof(hipComplex) * Td_k * Td_k);
    I    = (hipComplex *) malloc(sizeof(hipComplex) * Td_k * Td_k);
    X    = (hipComplex *) malloc(sizeof(hipComplex) * Td_k * Td_k);
    Hdag = (hipComplex *) malloc(sizeof(hipComplex) * M    * Td_k);
    W    = (hipComplex *) malloc(sizeof(hipComplex) * M);
    Ipiv = (int *) malloc(sizeof(int) * Td_k);
    info = (int *) malloc(sizeof(int));

    stopTime(&timer); printf("Host Allocs:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

    // set up device environment
    cudaError1 = hipStreamCreateWithFlags(&cudaStream, hipStreamNonBlocking);
    assert(hipSuccess == cudaError1);

    hipblasStatus_t = hipblasCreate(&cublasHandle);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

    hipblasStatus_t = hipblasSetStream(cublasHandle, cudaStream);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

    hipblasStatus_t = hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

    cusolverStatus = hipsolverDnCreate(&cusolverDnHandle);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolverStatus);

    cusolverStatus = hipsolverSetStream(cusolverDnHandle, cudaStream);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolverStatus);

    stopTime(&timer); printf("Startups:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

    // device allocations and parameter delivery
    // this is clearly why no one does this
    cudaError1 = hipMalloc((void**)&M_d,      sizeof(unsigned int));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&Td_k_d,   sizeof(unsigned int));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&Td_deg_d, sizeof(float) * 64);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&d_d,      sizeof(float));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&P_d,      sizeof(float));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&B_d,      sizeof(float));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&Td_c_d,   sizeof(float));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&alpha_d,  sizeof(hipComplex));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&beta_d,   sizeof(hipComplex));
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMalloc((void**)&scale_d,  sizeof(hipComplex));
    assert(hipSuccess == cudaError1);

    cudaError1 = hipMemcpyAsync(M_d, &M,          sizeof(unsigned int), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(Td_k_d, &Td_k,    sizeof(unsigned int), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(Td_deg_d, Td_deg, sizeof(float) * 64, hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(d_d, &d,          sizeof(float), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(P_d, &P,          sizeof(float), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(B_d, &B,          sizeof(float), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(Td_c_d, &Td_c,    sizeof(float), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(alpha_d, &alpha,  sizeof(hipComplex), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(beta_d, &beta,    sizeof(hipComplex), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);
    cudaError1 = hipMemcpyAsync(scale_d, &scale,  sizeof(hipComplex), hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);

    cudaError1 = hipMalloc((void**)&Td_d, sizeof(float) * 64);
    assert(hipSuccess == cudaError1);

    stopTime(&timer); printf("Dev Allocs1:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

    // calculate directions in radians (copy degrees, then scale vector)
    hipblasStatus_t = hipblasScopy(cublasHandle, Td_k, Td_deg_d, 1, Td_d, 1);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);
    hipblasStatus_t = hipblasSscal(cublasHandle, Td_k, Td_c_d, Td_d, 1);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

    stopTime(&timer); printf("Dev Td:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

    cudaError1 = hipMalloc((void**)&H_Td_d, sizeof(hipComplex) * M    * Td_k);
    cudaError2 = hipMalloc((void**)&HHLU_d, sizeof(hipComplex) * Td_k * Td_k);
    cudaError3 = hipMalloc((void**)&IX_d,   sizeof(hipComplex) * Td_k * Td_k);
    cudaError4 = hipMalloc((void**)&Hdag_d, sizeof(hipComplex) * M    * Td_k);
    assert(hipSuccess == cudaError1);
    assert(hipSuccess == cudaError2);
    assert(hipSuccess == cudaError3);
    assert(hipSuccess == cudaError4);
    cudaError1 = hipMalloc((void**)&s_d,    sizeof(hipComplex) * Td_k);
    cudaError2 = hipMalloc((void**)&W_d,    sizeof(hipComplex) * M);
    assert(hipSuccess == cudaError1);
    assert(hipSuccess == cudaError2);

    cudaError1 = hipMalloc((void**)&Ipiv_d, sizeof(int) * Td_k);
    cudaError2 = hipMalloc((void**)&info_d, sizeof(int));
    assert(hipSuccess == cudaError1);
    assert(hipSuccess == cudaError2);

    stopTime(&timer); printf("Dev Allocs 2:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

    // invoke kernel to construct steering matrix
    dim3 steeringgridDim((Td_k + STEERING_BLOCK_SIZE - 1) / STEERING_BLOCK_SIZE, (M + STEERING_BLOCK_SIZE - 1) / STEERING_BLOCK_SIZE, 1);
    dim3 steeringblockDim(STEERING_BLOCK_SIZE, STEERING_BLOCK_SIZE, 1);
    steering<<<steeringgridDim, steeringblockDim, 0, cudaStream>>>(M, Td_k, d, Td_d, H_Td_d);

    stopTime(&timer); printf("H_Td kernel:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

//    // this is for testing and intended to be temporary...
//    cudaError1 = cudaDeviceSynchronize();
//    assert(cudaSuccess == cudaError1);
//
//    cudaError1 = cudaMemcpy(H_Td, H_Td_d, sizeof(cuComplex) * M * Td_k, cudaMemcpyDeviceToHost);
//    assert(cudaSuccess == cudaError1);
//
//    printf("H =\n");
//    printMatrix(M, Td_k, H_Td, M, "H");
//    printf("=====\n");

// The next step is to form the matrix that will be inverted.  This
// matrix is formed by calculating dm_H_Td^H * dm_H_Td; that is, the
// the conjugate transpose of dm_H_Td is multiplied by dm_H_Td.
// This matrix multiplication is unusual because it can be done
// without explicitly forming the conjugate transpose; the only catch
// is that the matrix multiply is somewhat nonstandard, although it
// might be worth investigating whether options exist to form the
// conjugate transpose on the fly.  The resulting matrix was called
// dm_H_Td_H_Td originally, but for brevity is being shortened to
// something more along the lines of dm_HH_h.

    hipblasStatus_t = hipblasCgemm(cublasHandle, HIPBLAS_OP_C, HIPBLAS_OP_N, Td_k, Td_k, M, alpha_d, H_Td_d, M, H_Td_d, M, beta_d, HHLU_d, Td_k);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

    stopTime(&timer); printf("H'H Cgemm:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

//    // this is for testing and intended to be temporary...
//    cudaError1 = cudaDeviceSynchronize();
//    assert(cudaSuccess == cudaError1);
//
//    cudaError1 = cudaMemcpy(HH, HHLU_d, sizeof(cuComplex) * Td_k * Td_k, cudaMemcpyDeviceToHost);
//    assert(cudaSuccess == cudaError1);
//
//    printf("H'H =\n");
//    printMatrix(Td_k, Td_k, HH, Td_k, "H'H");
//    printf("=====\n");

// an inverse is now required, so do the setup to get that going

/* step 3: query working space of getrf */
    cusolverStatus = hipsolverDnCgetrf_bufferSize(
        cusolverDnHandle,
        Td_k,
        Td_k,
        HHLU_d,
        Td_k,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolverStatus);

    cudaError1 = hipMalloc((void**)&work_d, sizeof(hipComplex) * lwork);
    assert(hipSuccess == cudaError1);

/* step 4: LU factorization */
    cusolverStatus = hipsolverDnCgetrf(
        cusolverDnHandle,
        Td_k,
        Td_k,
        HHLU_d,
        Td_k,
        work_d,
        Ipiv_d,
        info_d);
//    cudaError1 = cudaDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolverStatus);
//    assert(cudaSuccess == cudaError1);

    stopTime(&timer); printf("LU:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

//    // async? still needed at all?
//    cudaError1 = cudaMemcpy(LU, HHLU_d, sizeof(cuComplex) * Td_k * Td_k, cudaMemcpyDeviceToHost);
//    cudaError2 = cudaMemcpy(Ipiv, Ipiv_d, sizeof(int) * Td_k, cudaMemcpyDeviceToHost);
//    cudaError3 = cudaMemcpy(info, info_d, sizeof(int), cudaMemcpyDeviceToHost);
//    assert(cudaSuccess == cudaError1);
//    assert(cudaSuccess == cudaError2);
//    assert(cudaSuccess == cudaError3);
//
//    if ( 0 > *info ){
//        printf("%d-th parameter is wrong \n", -*info);
//        exit(1);
//    }
//    printf("pivoting sequence, matlab base-1\n");
//    for(int j = 0 ; j < Td_k ; j++){
//        printf("Ipiv(%d) = %d\n", j+1, Ipiv[j]);
//    }
//    printf("L and U = (matlab base-1)\n");
//    printMatrix(Td_k, Td_k, LU, Td_k, "LU");
//    printf("=====\n");

// side step:  need an identiy matrix - was B before, will be X on device after solve
// so, also need a place for X to come back to on host

    // invoke kernel to construct complex identity matrix
    dim3 CidentgridDim((Td_k + CIDENT_BLOCK_SIZE - 1) / CIDENT_BLOCK_SIZE, (Td_k + CIDENT_BLOCK_SIZE - 1) / CIDENT_BLOCK_SIZE, 1);
    dim3 CidentblockDim(CIDENT_BLOCK_SIZE, CIDENT_BLOCK_SIZE, 1);
    Cident<<<CidentgridDim, CidentblockDim, 0, cudaStream>>>(Td_k, IX_d);

    stopTime(&timer); printf("I:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

/*
 * step 5: solve A*X = I
 * ????
 *
 */
    cusolverStatus = hipsolverDnCgetrs(
        cusolverDnHandle,
        HIPBLAS_OP_N,
        Td_k,
        Td_k, /* nrhs */
        HHLU_d, // this is now LU factored
        Td_k,
        Ipiv_d,
        IX_d,   // solution overwrites identity
        Td_k,
        info_d);
//    cudaError1 = cudaDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolverStatus);
//    assert(cudaSuccess == cudaError1);

    stopTime(&timer); printf("Inverse:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

//    cudaError1 = cudaMemcpy(X , IX_d, sizeof(cuComplex) * Td_k * Td_k, cudaMemcpyDeviceToHost);
//    assert(cudaSuccess == cudaError1);
//
//    printf("(H'H)^-1 = (matlab base-1)\n");
//    printMatrix(Td_k, Td_k, X, Td_k, "(H'H)^-1");
//    printf("=====\n");

// The next step is to form the pseudoinverse.  This matrix is formed
// by multiplying the steering matrix by the inverse just calculated.
// The steering matrix is generally not square, but the inverse is.
// They share inner dimensions so this is a regular matrix multiply.

    hipblasStatus_t = hipblasCgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, Td_k, Td_k, alpha_d, H_Td_d, M, IX_d, Td_k, beta_d, Hdag_d, M);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

    stopTime(&timer); printf("Pseudoinverse:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

//    // this is for testing and intended to be temporary...
//    cudaError1 = cudaDeviceSynchronize();
//    assert(cudaSuccess == cudaError1);
//
//    cudaError1 = cudaMemcpy(Hdag, Hdag_d, sizeof(cuComplex) * M * Td_k, cudaMemcpyDeviceToHost);
//    assert(cudaSuccess == cudaError1);
//
//    printf("Hdag =\n");
//    printMatrix(M, Td_k, Hdag, M, "Hdag");
//    printf("=====\n");

// The next step is to calculate the baseband weights, which is a
// matrix*vector multiplication.  The result also has to be scaled.

    s    = (hipComplex *) malloc(sizeof(hipComplex) * Td_k);
    s[0] = make_hipFloatComplex( 1., 0.);
    s[1] = make_hipFloatComplex(-1., 0.);

    cudaError1 = hipMemcpyAsync(s_d, s,          sizeof(hipComplex) * Td_k, hipMemcpyHostToDevice, cudaStream);
    assert(hipSuccess == cudaError1);

    hipblasStatus_t = hipblasCgemv(cublasHandle, HIPBLAS_OP_N, M, Td_k, scale_d, Hdag_d, M, s_d, 1, beta_d, W_d, 1);
    assert(HIPBLAS_STATUS_SUCCESS == hipblasStatus_t);

    stopTime(&timer); printf("W:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

    // this is for testing and intended to be temporary...
    cudaError1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaError1);

    cudaError1 = hipMemcpy(W, W_d, sizeof(hipComplex) * M, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaError1);

    stopTime(&timer); printf("W Retrieved:  %f s\n", elapsedTime(timer)); fflush(stdout); startTime(&timer);

    printf("W =\n");
    printMatrix(M, 1, W, M, "W");
    printf("=====\n");

    // free resources, shut down environment
//    if (dm_d  ) cudaFree(dm_d);
    if (Td_d  ) hipFree(Td_d);
    if (H_Td_d) hipFree(H_Td_d);
    if (HHLU_d) hipFree(HHLU_d);
    if (IX_d  ) hipFree(IX_d);
    if (work_d) hipFree(work_d);
    if (Hdag_d) hipFree(Hdag_d);
    if (s_d   ) hipFree(s_d);
    if (W_d   ) hipFree(W_d);
    if (Ipiv_d) hipFree(Ipiv_d);
    if (info_d) hipFree(info_d);

    if (cusolverDnHandle) hipsolverDnDestroy(cusolverDnHandle);
    if (cublasHandle)     hipblasDestroy(cublasHandle);
    if (cudaStream)       hipStreamDestroy(cudaStream);

    hipDeviceReset();

//    if (dm_h     ) free(dm_h     );
//    if (dm_Td    ) free(dm_Td    );
    if (H_Td) free(H_Td);
    if (HH  ) free(HH  );
    if (LU  ) free(LU  );
    if (I   ) free(I   );
    if (X   ) free(X   );
    if (Hdag) free(Hdag);
    if (s   ) free(s   );
    if (W   ) free(W   );
    if (Ipiv) free(Ipiv);
    if (info) free(info);

    return 0;
}

